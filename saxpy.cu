#include "hip/hip_runtime.h"
#include <stdio.h>

__global__

void saxpy(int n,float a, float *x,float *y)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i] = a*x[i]+y[i];
}

int main(void)
{
    int N =8<<20;
    float *x, *y,*d_x,*d_y;
    float r,s,t,e;

    printf("\nEnter the 3 val=");
    scanf("%f%f%f",&r,&s,&t);
    printf("\nEnter the error correction val=");
    scanf("%f",&e);

    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    for(int i =0;i<N;i++){
        x[i] = s;
        y[i]= t;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    //Perform Saxpy on 1M elements
    saxpy<<<(N+255)/256,256>>>(N, r, d_x,d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyHostToDevice);

    float maxError = 0.0f;
    for(int i=0;i<N;i++)
        maxError= max(maxError, abs(y[i]-e));
    printf("Max error:%f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}